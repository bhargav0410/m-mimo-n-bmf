#include "hip/hip_runtime.h"
#ifndef cudaEn
	#define cudaEn
#endif

//Shared Memory 
#include "ShMemSymBuff_cucomplex.hpp"
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#define FFT_size dimension
#define cp_size prefix
#define numSymbols lenOfBuffer

//gpu

#define threadsPerBlock FFT_size
#define numOfBlocks numOfRows

//LS
#define fileNameForX "Pilots.dat"
#define mode 0
/*
	mode:
		= 1 -> master -> creates shared memory 
		= 0 -> slave -> doesn't create the shared memory
*/
 
//!How to Compile:   nvcc ../../examples/gpuLS_cucomplex.cu -lcufft -lrt -o gpu 
// ./gpu

//LS
//Y = 16 x 1024
//X = 1 x 1023
//H = 16 x 1023
ShMemSymBuff* buffPtr;

using namespace std;

std::string file = "Output.dat";
//std::ofstream outfile;
	
	
//Reads in Vector X from file -> 1xcols
void matrix_readX(hipFloatComplex* X, int cols){
	ifstream inFile;
	inFile.open(fileNameForX);
	if (!inFile) {
		cerr << "Unable to open file "<< fileNameForX<<", filling in 1+i for x\n";
		float c=1.0f;
		for (int col = 0; col <  cols; col++){
			X[col].x=c;
			X[col].y=c;
		}
		return;
	}
	inFile.read((char*)X, (cols)*sizeof(*X));
	/*
	float c=0;
	for (int col = 0; col <  cols; col++){
		inFile >> c;
		X[col].real=c;
		inFile >> c;
		X[col].imag=c;
	}
	*/
	hipFloatComplex* temp = 0;
	temp=(hipFloatComplex*)malloc ((cols-1)/2* sizeof (*temp));
	//copy second half to temp
	memmove(temp, &X[(cols+1)/2], (cols-1)/2* sizeof (*X));
	//copy first half to second half
	memmove(&X[(cols-1)/2], X, (cols+1)/2* sizeof (*X));
	//copy temp to first half
	memmove(X, temp, (cols-1)/2* sizeof (*X));
	
	free(temp);
	inFile.close();
}

void shiftOneRow(hipFloatComplex* Y, int cols, int row){
	hipFloatComplex* Yf = &Y[row*cols];
	//std::cout << "Here...\n";
	hipFloatComplex* temp = 0;
	temp=(hipFloatComplex*)malloc ((cols+1)/2* sizeof (*temp));
	//copy second half to temp
	memmove(temp, &Yf[(cols-1)/2], (cols+1)/2* sizeof (*Yf));
	//copy first half to second half
	memmove(&Yf[(cols+1)/2], Yf, (cols-1)/2* sizeof (*Yf));
	//copy temp to first half
	memmove(Yf, temp, (cols+1)/2* sizeof (*Yf));
	
	free(temp);
	
}

void dropPrefix(hipFloatComplex *Y, hipFloatComplex *dY, int rows1, int cols1){
	
	int rows = rows1;
	int cols= cols1;	
	for(int i =0; i<rows; i++){
		memcpy(&Y[i*cols], &dY[i*(cols+prefix)+ prefix], cols*sizeof(*dY));
	}		
	
}

__global__ void findHs(hipFloatComplex* dY,hipFloatComplex* dH,hipFloatComplex* dX,int rows1,int cols1){
	
	//int rows = rows1;
	int cols=cols1;
	int rows = rows1;
	//find my work
	//Drop first element and copy it into Hconj
	for (int i = 0; i < rows; i++) {
		memcpy(&dH[i*(cols-1)], &dY[i*cols+1], (cols-1)* sizeof (*dY));
	}
	
	//complex division
	//H/X where H = FFT(Y) (w/ dropped first element)
	//Then take conjugate of H
	int i = blockIdx.x;
	int j = threadIdx.x;
	//for(int j=0; j<c; j++){
	if (j < cols) {
	//	dH[i*blockDim.x + j] = dY[i*blockDim.x + j + 1];
		dH[i*blockDim.x + j] = hipConjf(hipCdivf(dH[i*blockDim.x + j], dX[j]));
	}
	//}
	
	//Now dH holds conj H
}

void findDistSqrd(hipFloatComplex* H, float* Hsqrd, int rows, int cols){
	//initialize first row since Hsqrd currently holds X
	for (int j = 0; j<cols; j++){
		//|H|^2 = real^2 + imag^2
		//Sum of |H|^2 is summing all elements in col j
		Hsqrd[j] = hipCabsf(H[j])*hipCabsf(H[j]);
		//Hsqrd[j].y = 0;
	}
	
	for (int i = 1; i<rows; i++){  
		for (int j = 0; j<cols; j++){
			//|H|^2 = real^2 + imag^2
			//Sum of |H|^2 is summing all elements in col j
			Hsqrd[j] = Hsqrd[j] + hipCabsf(H[i*cols + j])*hipCabsf(H[i*cols + j]);
		}
	}
	
}

void firstVector(hipFloatComplex* dY, hipFloatComplex* dH, hipFloatComplex* dX, float* Hsqrd, int rows, int cols){
	
	//X = 1x1023 -> later can become |H|^2
	hipFloatComplex* X = 0;
	int sizeX=(cols-1)* sizeof(*X);
	X = (hipFloatComplex*)malloc(sizeX);
	//complexF* H =0;
	//H = (complexF *)malloc(sizeX*rows);
	//hipMalloc((void**)&H, size);
	
	//Read in X vector -> 1x1023
	matrix_readX(X, cols-1);
	hipMemcpy(dX, X, sizeX, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	
	
	// CUFFT plan -> do it one time before?
	
	
	//Read in Y with prefix
	buffPtr->readNextSymbol(dY, 0);
	decode[0]=0;
	//drop the prefix and move into first part of dY
	hipFloatComplex* Y = 0;
	hipMalloc((void**)&Y, rows*cols*sizeof(*Y));
	hipfftHandle plan;
    hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipMemcpy(Y, dY, rows*cols*sizeof(*Y), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	
	/*
	if(prefix>0){
		clock_t start, finish;
		if(timerEn){
			start = clock();
		}
		dropPrefix(Y, dY, rows, cols);
		hipDeviceSynchronize();
		if(timerEn){
			finish = clock();
			drop[0] = ((float)(finish - start))/(float)CLOCKS_PER_SEC;
		}
	}
	*/
	
		//hipfftExecC2C(plan, (hipfftComplex *)dY, (hipfftComplex *)dY, HIPFFT_FORWARD);
	
	clock_t start, finish;
	if(timerEn){
		start = clock();
	}
	
	
	//FFT(Y)
//	hipfftHandle plan;
//	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	/*
	int c = cols-1;
	for(int row=0; row<rows; row++){
		complexF* Yf = &dY[row*c];
		complexF* temp = 0;
		temp=(complexF*)malloc ((cols+1)/2* sizeof (*temp));
		//copy second half to temp
		memcpy(temp, &Yf[(c-1)/2], (c+1)/2* sizeof (*Yf));
		//copy first half to second half
		memcpy(&Yf[(c+1)/2], Yf, (c-1)/2* sizeof (*Yf));
		//copy temp to first half
		memcpy(Yf, temp, (c+1)/2* sizeof (*Yf));
		
		free(temp);
	}
	*/
	
	//find Hcon and Hsqrd
	findHs<< <numOfBlocks, threadsPerBlock-1 >> >(Y, dH, dX, rows, cols);
	hipDeviceSynchronize();
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	hipFloatComplex *H = 0;
	H = (hipFloatComplex*)malloc(rows*(cols-1)*sizeof(*H));
	hipMemcpy(H, dH, rows*(cols-1)*sizeof(*dH), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

	//H holds Hconj
	//hipMemcpy(H, dH, sizeX*rows, hipMemcpyDeviceToHost);
	
	//Save |H|^2 into X
	findDistSqrd(H,Hsqrd,rows, cols-1);
	
	std::string file = "Chan_est.dat";
	hipFloatComplex* Yf;
	Yf = (hipFloatComplex*)malloc(rows*(cols-1)*sizeof(*Yf));
	hipMemcpy(Yf, dH, rows*(cols-1)*sizeof(*dH), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	std::cout << "After Chan Est...\n";
	outfile.open(file.c_str(), std::ofstream::binary);
	outfile.write((const char*)Yf, rows*(cols-1)*sizeof(*Yf));
	outfile.close();
	/*
	memcpy(Yf, Hsqrd, (cols-1)*sizeof(*Hsqrd));
	std::cout << "After Squared...\n";
	file = "Dist_sqrd.dat";
	outfile.open(file.c_str(), std::ofstream::binary);
	outfile.write((const char*)Yf, (cols-1)*sizeof(*Yf));
	outfile.close();
	*/
	
	if(timerEn){
		finish = clock();
		decode[0] = decode[0]+ ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	
	free(X);
	free(Yf);
	free(H);
	
	//dH holds H conj
	//dX holds {H^2)	
}


__global__ void doOneSymbol(hipFloatComplex* Y, hipFloatComplex* Hconj, int rows1, int cols1){
	int rows = rows1;
	int cols= cols1;
    
    //find my work 
    int row = blockIdx.x;
	//printf("Row: %d\n",row);
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1023
	int i = row;
	int j = threadIdx.x;
	//int cp = cols;
	
	
	
	int c = cols-1;
	
	hipFloatComplex* Yf = 0;
	Yf = (hipFloatComplex*)malloc(rows*(cols-1)*sizeof(*Yf));
	/*
	extern __shared__ hipFloatComplex temp[];
	int tid = threadIdx.x;
	if (j > 0) {
		temp[tid] = Y[i*c + j];
	}
	__syncthreads();
	
	if (j < rows*(cols-1)) {
		Yf[i*c + j] = temp[tid];
		temp[tid].x = 0;
		temp[tid].y = 0;
	}
	*/
	/*
	for (int i = 0; i < rows; i++) {
		memcpy(&Yf[i*(cols-1)], &Y[i*cols+1], (cols-1)* sizeof (*Yf));
	}
	*/
	if (j < cols-1) {
		Yf[i*c + j] = Y[i*c + j + 1];
		Yf[i*c+j] = hipCmulf(Yf[i*c+j],Hconj[i*c+j]);
		Y[i*c + j] = Yf[i*c + j];
	}
	__syncthreads();
	free(Yf);
	//free(temp);
}

void symbolPreProcess(hipFloatComplex *Y, hipFloatComplex *Hconj, float *Hsqrd,int rows1, int cols1, int it) {
	int rows = rows1;
	int cols= cols1;	
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1023
	
	hipFloatComplex* dY = 0;
	hipMalloc((void**)&dY, rows*cols*sizeof(*dY));
	hipMemcpy(dY, Y, rows*cols*sizeof(*Y), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	std::cout << "Symbol " << it << ": " << hipGetErrorString(hipGetLastError()) << std::endl;

	
	if (it == 1) {
		std::string file = "Prefix_drop.dat";
		hipFloatComplex *Yf;
		Yf = (hipFloatComplex*)malloc(rows*cols*sizeof(*Yf));
		hipMemcpy(Yf, dY, rows*cols*sizeof(*Yf), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
		std::cout << "\n After Prefix drop:\n";
		for (int j = 0; j < rows*(cols); j = j + cols) {
			cout << "(" << Yf[j].x << ", " << Yf[j].y << ")\n";
		}
		
		outfile.open(file.c_str(), std::ofstream::binary);
		outfile.write((const char*)Yf, rows*(cols)*sizeof(*Yf));
		outfile.close();
		free(Yf);
	}
	
	clock_t start, finish;
	if(timerEn){
		start = clock();
	}
	
	//FFT(Y)
	hipfftHandle plan;
    hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)dY, (hipfftComplex *)dY, HIPFFT_FORWARD);
	
	
	if (it == 1) {
		std::string file = "FFT_Out.dat";
		hipFloatComplex* Yf;
		Yf = (hipFloatComplex*)malloc(rows*cols*sizeof(*Yf));
		hipMemcpy(Yf, dY, rows*cols*sizeof(*Yf), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
		std::cout << "After FFT...\n";
		outfile.open(file.c_str(), std::ofstream::binary);
		outfile.write((const char*)Yf, rows*(cols)*sizeof(*Yf));
		outfile.close();
		free(Yf);
	}
	
	doOneSymbol<< <numOfBlocks, threadsPerBlock>> >(dY, Hconj, rows, cols);
	hipDeviceSynchronize();
	hipMemcpy(Y, dY, rows*cols*sizeof(*Y), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	for(int r=1; r<rows; r++){
		for(int j=0; j<cols-1; j++){
			Y[j]= hipCaddf(Y[j],Y[r*cols+j]);
		}
	}
	
	//Divide YH* / |H|^2
	for(int j=0; j<cols-1; j++){
		Y[j].x = Y[j].x/Hsqrd[j];
		Y[j].y = Y[j].y/Hsqrd[j];
	}
	
	
	shiftOneRow(Y, cols-1, 0);
	
	if(timerEn){
		finish = clock();
		decode[it] = ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}	
	hipFree(dY);
	hipDeviceSynchronize();
}



int main(){
	int rows = numOfRows; // number of vectors
	int cols=dimension;//dimension
	hipSetDevice(0);
	printf("CUDA LS: \n");
	printInfo();
	//dY holds symbol with prefix
	hipFloatComplex *dY = 0;
	dY = (hipFloatComplex*)malloc(rows*cols* sizeof (*dY));
	
	float *Hsqrd = 0;
	Hsqrd = (float*)malloc((cols-1)*sizeof (*Hsqrd));
	
	//dH (and Hconj) = 16x1023
	hipFloatComplex *dH = 0;
	hipMalloc((void**)&dH, rows*(cols-1)* sizeof (*dH));
	
	//X = 1x1023 -> later can become |H|^2
	hipFloatComplex *dX = 0;
	hipMalloc((void**)&dX, (cols-1)* sizeof (*dX));
	
	hipFloatComplex *Yf = 0;
	Yf = (hipFloatComplex*)malloc((cols-1)* sizeof (*Yf));
	
	//Shared Memory
	string shm_uid = shmemID;
	buffPtr=new ShMemSymBuff(shm_uid, mode);
	
	/*
	hipfftComplex *temp = 0;
	hipMalloc((void**)&temp, rows*cols* sizeof (*temp));
	hipfftHandle plan;
    hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)&temp, (hipfftComplex *)&temp, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	*/
	
	//Find H* (H conjugate) ->16x1023 and |H|^2 -> 1x1023
	firstVector(dY, dH, dX, Hsqrd, rows, cols);
	//dH holds h conj
	//dX holds |H|^2
	
	for(int i=1; i<numberOfSymbolsToTest; i++){
		if(i==numberOfSymbolsToTest-1){
			//if last one
			buffPtr->readLastSymbol(dY);
		}
		else{
			buffPtr->readNextSymbol(dY,i);
			
			if (i == 1) {
				std::string file = "Sym_copy.dat";
//				hipFloatComplex Yf_[rows*(cols+prefix)];
				hipFloatComplex *Yf_;
				Yf_ = (hipFloatComplex*)malloc(rows*(cols)*sizeof(*Yf_));
				memcpy(Yf_, dY, rows*(cols)*sizeof(*Yf_));
				//hipDeviceSynchronize();
				//std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
				std::cout << "Copied back to CPU...\n";
				//printOutArr(Yf_,1,cols+prefix);
					for (int j = 0; j < rows*(cols); j = j + cols) {
						cout << "(" << Yf_[j].x << ", " << Yf_[j].y << ")\n";
					}
				outfile.open(file.c_str(), std::ofstream::binary);
				outfile.write((const char*)Yf_, rows*(cols)*sizeof(*Yf_));
				outfile.close();
			}
			
		}
		symbolPreProcess(dY, dH, Hsqrd, rows, cols, i);
		
		if(testEn){
			//printf("Symbol #%d:\n", i);
			//cuda copy it over
			memcpy(Yf, dY, (cols-1)* sizeof (*Yf));
			if (i <= 1) {
				outfile.open(file.c_str(), std::ofstream::binary | std::ofstream::trunc);
			} else {
				outfile.open(file.c_str(), std::ofstream::binary | std::ofstream::app);
			}
			outfile.write((const char*)Yf, (cols-1)*sizeof(*Yf));
			outfile.close();
			//printOutArr(Yf, 1, cols-1);
		}
		
		
	}
	
	free(Yf);
	hipFree(dY);
	hipFree(dH);
	hipFree(dX);
	//delete buffPtr;
	if(timerEn)
		printTimes(true);
	return 0;

}