#include "hip/hip_runtime.h"
#ifndef cudaEn
	#define cudaEn
#endif

//Shared Memory 
#include "gpuLS.cuh"
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>
#include <csignal>
#define FFT_size dimension
#define cp_size prefix
#define numSymbols lenOfBuffer


/*
	mode:
		= 1 -> master -> creates shared memory 
		= 0 -> slave -> doesn't create the shared memory
*/
 
//!How to Compile:   nvcc ../../examples/gpuLS_full_frame.cu -lcufft -lrt -o gpu -arch=sm_35
// ./gpu

//LS
//Y = 16 x 1024
//X = 1 x 1023
//H = 16 x 1023
using namespace std;
int device_number = 0;

static bool stop_signal_called = false;
void sig_int_handler(int){stop_signal_called = true;}

std::string file = "Output_gpu.dat";
std::ofstream outfile;

int main(){
	gpuLS *gpu  = new gpuLS;
	
	int rows = numOfRows; // number of vectors
	int cols=dimension;//dimension
	device_number = 0;
	hipSetDevice(device_number);
	hipGetDeviceProperties(&gpu->devProp, device_number);
	
	//dY holds symbol with prefix
	hipFloatComplex *dY = 0;
	dY = (hipFloatComplex*)malloc(rows*(cols)*lenOfBuffer* sizeof (*dY));
	
	float *Hsqrd = 0;
	hipMalloc((void**)&Hsqrd, (cols-1)* sizeof (*Hsqrd));
	
	//dH (and Hconj) = 16x1023
	hipFloatComplex *dH = 0;
	hipMalloc((void**)&dH, rows*(cols-1)* sizeof (*dH));
	
	//X = 1x1023 -> later can become |H|^2
	hipFloatComplex *dX = 0;
	hipMalloc((void**)&dX, rows*(cols-1)* sizeof (*dX));
	
	hipFloatComplex *Yf = 0;
	Yf = (hipFloatComplex*)malloc((cols-1)*(lenOfBuffer-1)* sizeof (*Yf));
	
	hipFloatComplex* Y = 0;
	hipMalloc((void**)&Y, rows*cols*lenOfBuffer*sizeof(*Y));
	
	clock_t start, finish;
	float frameTime;
	
	hipfftHandle plan;
	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipDeviceSynchronize();

	std::signal(SIGINT, &sig_int_handler);
	
	gpu->copyPilotToGPU(dX, rows, cols);
	
//	while (not stop_signal_called) {
		start = clock();
		
		for (int it = 0; it < numberOfSymbolsToTest; it++) {
			if(it==numberOfSymbolsToTest-1){
				//if last one
				gpu->buffPtr->readLastSymbolCUDA(&Y[rows*cols*it]);
			} else {
				gpu->buffPtr->readNextSymbolCUDA(&Y[rows*cols*it], it);
			}
		}
		hipDeviceSynchronize();
		gpu->demodOneFrameCUDA(dY, Y, dX, dH, Hsqrd, rows, cols);
		if(timerEn) {
			gpu->buffPtr->printTimes(true);
			gpu->buffPtr->storeTimes(false);
		}
		if(testEn){
			//printf("Symbol #%d:\n", i);
			//cuda copy it over
			memcpy(Yf, dY, (cols-1)*(lenOfBuffer-1)* sizeof (*Yf));
			outfile.open(file.c_str(), std::ofstream::binary | std::ofstream::trunc);
			outfile.write((const char*)Yf, (cols-1)*(lenOfBuffer-1)*sizeof(*Yf));
			outfile.close();
		}
		/*
		hipDeviceReset();
		hipMalloc((void**)&Hsqrd, (cols-1)* sizeof (*Hsqrd));
		hipMalloc((void**)&dH, rows*(cols-1)* sizeof (*dH));
		hipMalloc((void**)&dX, rows*(cols-1)* sizeof (*dX));
		hipMalloc((void**)&Y, rows*cols*lenOfBuffer*sizeof(*Y));
		*/
		while ((((float)(clock() - start))/(float)CLOCKS_PER_SEC) < 1);
//	}
	
	free(Yf);
	free(dY);
	hipFree(Y);
	hipFree(dH);
	hipFree(dX);
	hipFree(Hsqrd);
	delete(gpu);
	hipDeviceReset();
	return 0;

}