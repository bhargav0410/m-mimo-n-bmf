#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"

#include "hip/hip_complex.h"

using namespace std;
#define N 1023

struct complexF{
	float real;
	float imag;
};

__global__ void addNums(hipFloatComplex *a, hipFloatComplex *b, hipFloatComplex *c) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < N)
		c[id] = hipCaddf(a[id],b[id]);
}

int main() {
	int count;
	hipGetDeviceCount(&count);
	cout << "\n\nThe number of devices supported are " << count << endl;
	hipDeviceProp_t devProp;
	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&devProp, i);
		cout << "Device ID: " << devProp.name << endl;
		cout << "Total global memory: " << devProp.totalGlobalMem << endl;
		cout << "Memory pitch: " << devProp.memPitch << endl;
		cout << "Total constant memory: " << devProp.totalConstMem << endl;
		cout << "Number of Processor(s): " << devProp.multiProcessorCount << endl;
		//cout << "Number of register(s) per processor: " << devProp.regsPerMultiprocessor << endl;
		cout << "Number of thread(s) per processor: " << devProp.maxThreadsPerMultiProcessor << endl;
		cout << "Number of thread(s) per block: " << devProp.maxThreadsPerBlock << endl;
	}
	

	hipFloatComplex a[N], b[N], ans[N];
	hipFloatComplex *dev_a, *dev_b, *c;
	hipMalloc((void**)&dev_a, N*sizeof(*dev_a));
	hipMalloc((void**)&dev_b, N*sizeof(*dev_b));
	hipMalloc((void**)&c, N*sizeof(*c));
	for (int i = 0; i < N; i++) {
		a[i].x = i;
		a[i].y = i * 3;
		b[i].x = i;
		b[i].y = i * 3;
	}
	hipMemcpy(dev_a, &a, N * sizeof(*dev_a), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, N * sizeof(*dev_b), hipMemcpyHostToDevice);
	addNums <<<N,N>>>(dev_a, dev_b, c);
	hipMemcpy(ans, c, N * sizeof(*c), hipMemcpyDeviceToHost);
	cout << "\nThe answer is ";
	for (int i = 0; i < N; i = i + 100) {
		cout << "\n" << a[i].x << "+" << b[i].x << "=" << ans[i].x << ", ";
		cout << "\n" << a[i].y << "+" << b[i].y << "=" << ans[i].y << ", ";
	}
	return 0;
}