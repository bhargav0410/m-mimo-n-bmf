#include "hip/hip_runtime.h"
#ifndef cudaEn
	#define cudaEn
#endif

//Shared Memory 
#include "ShMemSymBuff_cucomplex.hpp"
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>
#define FFT_size dimension
#define cp_size prefix
#define numSymbols lenOfBuffer

//gpu

#define threadsPerBlock FFT_size
#define numOfBlocks numOfRows

//LS
#define fileNameForX "Pilots.dat"
#define mode 0
/*
	mode:
		= 1 -> master -> creates shared memory 
		= 0 -> slave -> doesn't create the shared memory
*/
 
//!How to Compile:   nvcc ../../examples/gpuLS_cucomplex.cu -lcufft -lrt -o gpu 
// ./gpu

//LS
//Y = 16 x 1024
//X = 1 x 1023
//H = 16 x 1023
ShMemSymBuff* buffPtr;

using namespace std;

std::string file = "Output_gpu.dat";
//std::ofstream outfile;
	
	
//Reads in Vector X from file -> 1xcols
void matrix_readX(hipFloatComplex* X, int cols){
	ifstream inFile;
	inFile.open(fileNameForX);
	if (!inFile) {
		cerr << "Unable to open file "<< fileNameForX<<", filling in 1+i for x\n";
		float c=1.0f;
		for (int col = 0; col <  cols; col++){
			X[col].x=c;
			X[col].y=c;
		}
		return;
	}
	inFile.read((char*)X, (cols)*sizeof(*X));
	/*
	float c=0;
	for (int col = 0; col <  cols; col++){
		inFile >> c;
		X[col].real=c;
		inFile >> c;
		X[col].imag=c;
	}
	*/
	hipFloatComplex* temp = 0;
	temp=(hipFloatComplex*)malloc ((cols-1)/2* sizeof (*temp));
	//copy second half to temp
	memmove(temp, &X[(cols+1)/2], (cols-1)/2* sizeof (*X));
	//copy first half to second half
	memmove(&X[(cols-1)/2], X, (cols+1)/2* sizeof (*X));
	//copy temp to first half
	memmove(X, temp, (cols-1)/2* sizeof (*X));
	
	free(temp);
	inFile.close();
}

void shiftOneRow(hipFloatComplex* Y, int cols, int row){
	hipFloatComplex* Yf = &Y[row*cols];
	//std::cout << "Here...\n";
	hipFloatComplex* temp = 0;
	temp=(hipFloatComplex*)malloc ((cols+1)/2* sizeof (*temp));
	//copy second half to temp
	memmove(temp, &Yf[(cols-1)/2], (cols+1)/2* sizeof (*Yf));
	//copy first half to second half
	memmove(&Yf[(cols+1)/2], Yf, (cols-1)/2* sizeof (*Yf));
	//copy temp to first half
	memmove(Yf, temp, (cols+1)/2* sizeof (*Yf));
	
	free(temp);
	
}

void dropPrefix(hipFloatComplex *Y, hipFloatComplex *dY, int rows1, int cols1){
	
	int rows = rows1;
	int cols= cols1;	
	for(int i =0; i<rows; i++){
		memcpy(&Y[i*cols], &dY[i*(cols+prefix)+ prefix], cols*sizeof(*dY));
	}		
}

__device__ hipFloatComplex tempDev[numOfBlocks*(threadsPerBlock-1)];
__global__ void reshapeSym(hipFloatComplex* Y, hipFloatComplex* Yf){
	int row = blockIdx.x;
	int col = threadIdx.x;
	tempDev[row*blockDim.x + col] = Yf[col*blockDim.x + row];
	__syncthreads();
	Y[row*blockDim.x + col] = tempDev[row*blockDim.x + col];
	__syncthreads();	
}

__global__ void findHs(hipFloatComplex* dY,hipFloatComplex* dH,hipFloatComplex* dX,int rows1,int cols1){	
	int cols=cols1;
	//find my work
	//Drop first element and copy it into Hconj
	dH[blockIdx.x*blockDim.x + threadIdx.x] = dY[blockIdx.x*(blockDim.x + 1) + threadIdx.x + 1];
	__syncthreads();
	
	//complex division
	//H/X where H = FFT(Y) (w/ dropped first element)
	//Then take conjugate of H
	if (threadIdx.x < (cols-1)) {
		dH[blockIdx.x*blockDim.x + threadIdx.x] = hipCdivf(dH[blockIdx.x*blockDim.x + threadIdx.x], dX[blockIdx.x*blockDim.x + threadIdx.x]);
		dH[blockIdx.x*blockDim.x + threadIdx.x] = hipConjf(dH[blockIdx.x*blockDim.x + threadIdx.x]);
		dX[blockIdx.x*blockDim.x + threadIdx.x].x = dH[blockIdx.x*blockDim.x + threadIdx.x].x * dH[blockIdx.x*blockDim.x + threadIdx.x].x + dH[blockIdx.x*blockDim.x + threadIdx.x].y * dH[blockIdx.x*blockDim.x + threadIdx.x].y;
	}
	__syncthreads();
	//Now dH holds conj H
	
	
}


__global__ void findDistSqrd(hipFloatComplex* H, float* Hsqrd, int rows, int cols){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	extern __shared__ hipFloatComplex temp[];
	int sid = threadIdx.x*cols + blockIdx.x;
	temp[threadIdx.x] = H[sid];
	//temp[col].x = temp[col].x*temp[col].x + temp[col].y*temp[col].y;
//	temp[threadIdx.x].x = tempLocal.x * tempLocal.x + tempLocal.y * tempLocal.y;
//	__syncthreads();
	
	
	for (int i = 1; i < rows; i = i*2) {
		if (threadIdx.x%(2*i) == 0) {
			temp[threadIdx.x].x += temp[threadIdx.x+i].x;
		}
		__syncthreads();
	}
	
	
	if(threadIdx.x == 0) {
		Hsqrd[blockIdx.x] = temp[threadIdx.x].x;
	/*	
		Hsqrd[blockIdx.x] = 0;
		for (int i = 0; i < numOfBlocks; i++) {
			
			Hsqrd[blockIdx.x] = Hsqrd[blockIdx.x] + H[i*cols + blockIdx.x].x;
		}
		*/
	}
}


void firstVector(hipFloatComplex* dY, hipFloatComplex* dH, hipFloatComplex* dX, float* Hsqrd, int rows, int cols){
	clock_t start, finish;
	//std::cout << "Here...\n";
	//X = 1x1023 -> later can become |H|^2
	hipFloatComplex* X = 0;
	int sizeX=rows*(cols-1)* sizeof(*X);
	X = (hipFloatComplex*)malloc(sizeX);
	//complexF* H =0;
	//H = (complexF *)malloc(sizeX*rows);
	//hipMalloc((void**)&H, size);
	
	//Read in X vector -> 1x1023
	for (int i = 0; i < rows; i++) {
		//std::cout << "Here...\n";
		matrix_readX(&X[i*(cols-1)], cols-1);
	}
	//std::cout << "Here...\n";
	hipMemcpy(dX, X, rows*(cols-1)*sizeof(*dX), hipMemcpyHostToDevice);
	hipDeviceSynchronize();	
	
	// CUFFT plan -> do it one time before?
	
	
	//Read in Y with prefix
	buffPtr->readNextSymbol(dY, 0);
	decode[0]=0;
	//drop the prefix and move into first part of dY
	hipFloatComplex* Y = 0;
	hipMalloc((void**)&Y, rows*cols*sizeof(*Y));
	hipfftHandle plan;
    hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	if(timerEn){
		start = clock();
	}
	hipMemcpy(Y, dY, rows*cols*sizeof(*Y), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		readT[0] = readT[0] + ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}

	
	if(timerEn){
		start = clock();
	}
	
	
	//FFT(Y)
//	hipfftHandle plan;
//	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		fft[0] = fft[0]+ ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	
	//find Hconj and Hsqrd
	hipFloatComplex *H = 0;
	hipMalloc((void**)&H, rows*(cols-1)* sizeof (*H));
	if(timerEn){
		start = clock();
	}
//	dim3 dimBlock(numOfBlocks, threadsPerBlock-1);
	findHs<< <numOfBlocks, threadsPerBlock-1>> >(Y, dH, dX, rows, cols);
	hipDeviceSynchronize();
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
//	reshapeSym<< <numOfBlocks,threadsPerBlock-1 >> >(H, dH);
//	hipDeviceSynchronize();
	//Save |H|^2 into Hsqrd
	findDistSqrd<< <threadsPerBlock-1, numOfBlocks, numOfBlocks*sizeof(hipFloatComplex)>> >(dX,Hsqrd,rows, cols-1);
	hipDeviceSynchronize();
	
	if(timerEn){
		finish = clock();
		decode[0] = decode[0] + ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	
	std::string file = "Chan_est.dat";
	hipFloatComplex* Yf;
	Yf = (hipFloatComplex*)malloc(rows*(cols-1)*sizeof(*Yf));
	hipMemcpy(Yf, dH, rows*(cols-1)*sizeof(*dH), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	std::cout << "After Chan Est...\n";
	outfile.open(file.c_str(), std::ofstream::binary);
	outfile.write((const char*)Yf, rows*(cols-1)*sizeof(*Yf));
	outfile.close();
	hipDeviceSynchronize();
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	
	hipMemcpy(Yf, Hsqrd, (cols-1)*sizeof(*Hsqrd), hipMemcpyDeviceToHost);
	std::cout << "After Squared...\n";
	file = "Dist_sqrd.dat";
	outfile.open(file.c_str(), std::ofstream::binary);
	outfile.write((const char*)Yf, rows*(cols-1)*sizeof(*Hsqrd));
	outfile.close();
	
	free(X);
	hipFree(H);
	
	//dH holds H conj
	//dX holds {H^2)	
}


__global__ void doOneSymbol(hipFloatComplex* Y, hipFloatComplex* Hconj, hipFloatComplex* Yf, int rows1, int cols1){
	int rows = rows1;
	int cols= cols1;
    
    //find my work 
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1023
	int row = blockIdx.x;
	int j = threadIdx.x;
	Yf[row*blockDim.x + j] = Y[row*(blockDim.x+1) + j + 1];
	__syncthreads();
	
	if (j < cols-1) {
		Yf[row*blockDim.x + j] = hipCmulf(Yf[row*blockDim.x + j],Hconj[row*blockDim.x + j]);
	}
	__syncthreads();
}


__global__ void combineForMRC(hipFloatComplex *Y, float *Hsqrd, int rows, int cols) {
	
	int row = blockIdx.x;
	int col = threadIdx.x;
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	extern __shared__ hipFloatComplex temp[];
	int sid = threadIdx.x*cols + blockIdx.x;
	temp[col] = Y[sid];
	
	for (int i = 1; i < rows; i = i*2) {
		if (threadIdx.x%(2*i) == 0) {
			temp[col] = hipCaddf(temp[col],temp[col+i]);
		}
		__syncthreads();
	}
	
	if (col == 0) {
		Y[row].x = temp[col].x/Hsqrd[row];
		Y[row].y = temp[col].y/Hsqrd[row];
		__syncthreads();
	}
	
	/*
	if (threadIdx.x == 0) {
		for (int i = 1; i < numOfRows; i++) {
			Y[blockIdx.x] = hipCaddf(Y[blockIdx.x],Y[i*cols + blockIdx.x]);
		}
		Y[blockIdx.x].x = Y[blockIdx.x].x/Hsqrd[blockIdx.x];
		Y[blockIdx.x].y = Y[blockIdx.x].y/Hsqrd[blockIdx.x];
	}
	*/
	
}


void symbolPreProcess(hipFloatComplex *Y, hipFloatComplex *Hconj, float *Hsqrd,int rows1, int cols1, int it) {
	int rows = rows1;
	int cols= cols1;

	clock_t start, finish;
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1023
	if(timerEn){
		start = clock();
	}
	
	hipFloatComplex* dY = 0;
	hipMalloc((void**)&dY, rows*cols*sizeof(*dY));
	hipMemcpy(dY, Y, rows*cols*sizeof(*dY), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	if(timerEn){
		finish = clock();
		readT[it] = readT[it] + ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	
	if(timerEn){
		start = clock();
	}
	
	//FFT(Y)
	hipfftHandle plan;
    hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)dY, (hipfftComplex *)dY, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		fft[it] = fft[it]+ ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	
	hipFloatComplex* Yf = 0;
	hipMalloc((void**)&Yf, rows*(cols-1)* sizeof (*Yf));
	
	if(timerEn){
		start = clock();
	}
	doOneSymbol<< <numOfBlocks, threadsPerBlock-1>> >(dY, Hconj, Yf, rows, cols);
	hipDeviceSynchronize();
	combineForMRC<< <threadsPerBlock-1, numOfBlocks, numOfBlocks*sizeof(hipFloatComplex)>> >(Yf, Hsqrd, rows, cols-1);
	hipMemcpy(Y, Yf, rows*(cols-1)*sizeof(*Y), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	/*
	if(timerEn){
		start = clock();
	}
	*/
	/*
	for(int r=1; r<rows; r++){
		for(int j=0; j<cols-1; j++){
			Y[j]= hipCaddf(Y[j],Y[r*(cols-1)+j]);
		}
	}
	
	//Divide YH* / |H|^2
	for(int j=0; j<cols-1; j++){
		Y[j].x = Y[j].x/Hsqrd[j];
		Y[j].y = Y[j].y/Hsqrd[j];
	}
	*/
	
	shiftOneRow(Y, cols-1, 0);
	
	if(timerEn){
		finish = clock();
		decode[it] = ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	hipFree(dY);
	hipFree(Yf);
	hipDeviceSynchronize();
}



int main(){
	int rows = numOfRows; // number of vectors
	int cols=dimension;//dimension
	hipSetDevice(0);
	//printf("CUDA LS: \n");
	//printInfo();
	//dY holds symbol with prefix
	hipFloatComplex *dY = 0;
	dY = (hipFloatComplex*)malloc(rows*cols* sizeof (*dY));
	
	float *Hsqrd = 0;
	hipMalloc((void**)&Hsqrd, (cols-1)* sizeof (*Hsqrd));
	
	//dH (and Hconj) = 16x1023
	hipFloatComplex *dH = 0;
	hipMalloc((void**)&dH, rows*(cols-1)* sizeof (*dH));
	
	//X = 1x1023 -> later can become |H|^2
	hipFloatComplex *dX = 0;
	hipMalloc((void**)&dX, rows*(cols-1)* sizeof (*dX));
	
	hipFloatComplex *Yf = 0;
	Yf = (hipFloatComplex*)malloc((cols-1)* sizeof (*Yf));
	
	//Shared Memory
	string shm_uid = shmemID;
	buffPtr=new ShMemSymBuff(shm_uid, mode);
	
	
	/*hipfftComplex *temp = 0;
	hipMalloc((void**)&temp, rows*cols* sizeof (*temp));
	hipfftHandle plan;
    hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)&temp, (hipfftComplex *)&temp, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	hipFree(temp);
	*/
	//Find H* (H conjugate) ->16x1023 and |H|^2 -> 1x1023
	firstVector(dY, dH, dX, Hsqrd, rows, cols);
	//dH holds h conj
	//dX holds |H|^2
	
	for(int i=1; i<numberOfSymbolsToTest; i++){
		if(i==numberOfSymbolsToTest-1){
			//if last one
			buffPtr->readLastSymbol(dY);
		}
		else{
			buffPtr->readNextSymbol(dY,i);
			/*
			if (i == 1) {
				std::string file = "Sym_copy.dat";
//				hipFloatComplex Yf_[rows*(cols+prefix)];
				hipFloatComplex *Yf_;
				Yf_ = (hipFloatComplex*)malloc(rows*(cols)*sizeof(*Yf_));
				memcpy(Yf_, dY, rows*(cols)*sizeof(*Yf_));
				//hipDeviceSynchronize();
				//std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
				std::cout << "Copied back to CPU...\n";
				//printOutArr(Yf_,1,cols+prefix);
					for (int j = 0; j < rows*(cols); j = j + cols) {
						cout << "(" << Yf_[j].x << ", " << Yf_[j].y << ")\n";
					}
				outfile.open(file.c_str(), std::ofstream::binary);
				outfile.write((const char*)Yf_, rows*(cols)*sizeof(*Yf_));
				outfile.close();
			}
			*/
			
		}
		symbolPreProcess(dY, dH, Hsqrd, rows, cols, i);
		
		if(testEn){
			//printf("Symbol #%d:\n", i);
			//cuda copy it over
			memcpy(Yf, dY, (cols-1)* sizeof (*Yf));
			if (i <= 1) {
				outfile.open(file.c_str(), std::ofstream::binary | std::ofstream::trunc);
			} else {
				outfile.open(file.c_str(), std::ofstream::binary | std::ofstream::app);
			}
			outfile.write((const char*)Yf, (cols-1)*sizeof(*Yf));
			outfile.close();
			//printOutArr(Yf, 1, cols-1);
		}
		
		
	}
	
	free(Yf);
	hipFree(dY);
	hipFree(dH);
	hipFree(dX);
	//delete buffPtr;
	
	if(timerEn) {
		printTimes(true);
		storeTimes(false);
	}
	return 0;

}