#include "hip/hip_runtime.h"
#ifndef cudaEn
	#define cudaEn
#endif

//Shared Memory 
#include "ShMemSymBuff_cucomplex.hpp"
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>
#define FFT_size dimension
#define cp_size prefix
#define numSymbols lenOfBuffer

//gpu

#define threadsPerBlock FFT_size
#define numOfBlocks numOfRows

//LS
#define fileNameForX "Pilots.dat"
#define mode 0
/*
	mode:
		= 1 -> master -> creates shared memory 
		= 0 -> slave -> doesn't create the shared memory
*/
 
//!How to Compile:   nvcc ../../examples/gpuLS_cucomplex.cu -lcufft -lrt -o gpu 
// ./gpu

//LS
//Y = 16 x 1024
//X = 1 x 1023
//H = 16 x 1023
ShMemSymBuff* buffPtr;

using namespace std;

std::string file = "Output_gpu.dat";
//std::ofstream outfile;
	
	
//Reads in Vector X from file -> 1xcols
void matrix_readX(hipFloatComplex* X, int cols){
	ifstream inFile;
	inFile.open(fileNameForX);
	if (!inFile) {
		cerr << "Unable to open file "<< fileNameForX<<", filling in 1+i for x\n";
		float c=1.0f;
		for (int col = 0; col <  cols; col++){
			X[col].x=c;
			X[col].y=c;
		}
		return;
	}
	inFile.read((char*)X, (cols)*sizeof(*X));
	/*
	float c=0;
	for (int col = 0; col <  cols; col++){
		inFile >> c;
		X[col].real=c;
		inFile >> c;
		X[col].imag=c;
	}
	*/
	hipFloatComplex* temp = 0;
	temp=(hipFloatComplex*)malloc ((cols-1)/2* sizeof (*temp));
	//copy second half to temp
	memmove(temp, &X[(cols+1)/2], (cols-1)/2* sizeof (*X));
	//copy first half to second half
	memmove(&X[(cols-1)/2], X, (cols+1)/2* sizeof (*X));
	//copy temp to first half
	memmove(X, temp, (cols-1)/2* sizeof (*X));
	
	free(temp);
	inFile.close();
}

void shiftOneRow(hipFloatComplex* Y, int cols, int row){
	hipFloatComplex* Yf = &Y[row*cols];
	//std::cout << "Here...\n";
	hipFloatComplex* temp = 0;
	temp=(hipFloatComplex*)malloc ((cols+1)/2* sizeof (*temp));
	//copy second half to temp
	memmove(temp, &Yf[(cols-1)/2], (cols+1)/2* sizeof (*Yf));
	//copy first half to second half
	memmove(&Yf[(cols+1)/2], Yf, (cols-1)/2* sizeof (*Yf));
	//copy temp to first half
	memmove(Yf, temp, (cols+1)/2* sizeof (*Yf));
	
	free(temp);
	
}

void dropPrefix(hipFloatComplex *Y, hipFloatComplex *dY, int rows1, int cols1){
	
	int rows = rows1;
	int cols= cols1;	
	for(int i =0; i<rows; i++){
		memcpy(&Y[i*cols], &dY[i*(cols+prefix)+ prefix], cols*sizeof(*dY));
	}		
	
}

__global__ void findHs(hipFloatComplex* dY,hipFloatComplex* dH,hipFloatComplex* dX,int rows1,int cols1){
	
	int cols=cols1;
	int rows = rows1;
	int row = blockIdx.x;
	int j = threadIdx.x;
	//Drop first element and copy it into Hconj
	__shared__ hipFloatComplex temp[threadsPerBlock-1];
	temp[j] = dY[row*(blockDim.x+1) + j + 1];
	__syncthreads();
	dH[row*blockDim.x + j] = temp[j];
	__syncthreads();
	//complex division
	//H/X where H = FFT(Y) (w/ dropped first element)
	//Then take conjugate of H
	if (j < (cols-1)) {
		//dH[i*blockDim.x + j] = dY[i*blockDim.x + j + 1];
		dH[row*blockDim.x + j] = hipCdivf(dH[row*blockDim.x + j], dX[row*blockDim.x + j]);
		dH[row*blockDim.x + j] = hipConjf(dH[row*blockDim.x + j]);
	}
	__syncthreads();
}

void findDistSqrd(hipFloatComplex* H, float* Hsqrd, int rows, int cols){
	//initialize first row since Hsqrd currently holds X
	for (int j = 0; j<cols; j++){
		Hsqrd[j] = 0;
		//|H|^2 = real^2 + imag^2
		//Sum of |H|^2 is summing all elements in col j
		Hsqrd[j] = H[j].x*H[j].x + H[j].y*H[j].y;
		//Hsqrd[j].y = 0;
	}
	
	for (int i = 1; i<rows; i++){  
		for (int j = 0; j<cols; j++){
			//|H|^2 = real^2 + imag^2
			//Sum of |H|^2 is summing all elements in col j
			Hsqrd[j] = Hsqrd[j] + (H[i*cols + j].x*H[i*cols + j].x + H[i*cols + j].y+H[i*cols + j].y);
		}
	}
	
}

void firstVector(hipFloatComplex* dY, hipFloatComplex* dH, hipFloatComplex* dX, float* Hsqrd, int rows, int cols){
	//std::cout << "Here...\n";
	//X = 1x1023 -> later can become |H|^2
	hipFloatComplex* X = 0;
	int sizeX=rows*(cols-1)* sizeof(*X);
	X = (hipFloatComplex*)malloc(sizeX);
	//complexF* H =0;
	//H = (complexF *)malloc(sizeX*rows);
	//hipMalloc((void**)&H, size);
	
	//Read in X vector -> 1x1023
	for (int i = 0; i < rows; i++) {
		//std::cout << "Here...\n";
		matrix_readX(&X[i*(cols-1)], cols-1);
	}
	//std::cout << "Here...\n";
	hipMemcpy(dX, X, rows*(cols-1)*sizeof(*dX), hipMemcpyHostToDevice);
	hipDeviceSynchronize();	
	
	// CUFFT plan -> do it one time before?
	
	
	//Read in Y with prefix
	buffPtr->readNextSymbolCUDA(dY, 0);
	decode[0]=0;
	//drop the prefix and move into first part of dY
	hipFloatComplex* Y = 0;
	hipMalloc((void**)&Y, rows*cols*sizeof(*Y));
	hipfftHandle plan;
    hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipMemcpy(Y, dY, rows*cols*sizeof(*Y), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
//	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	
	/*
	if(prefix>0){
		clock_t start, finish;
		if(timerEn){
			start = clock();
		}
		dropPrefix(Y, dY, rows, cols);
		hipDeviceSynchronize();
		if(timerEn){
			finish = clock();
			drop[0] = ((float)(finish - start))/(float)CLOCKS_PER_SEC;
		}
	}
	*/
	
		//hipfftExecC2C(plan, (hipfftComplex *)dY, (hipfftComplex *)dY, HIPFFT_FORWARD);
	
	clock_t start, finish;
	if(timerEn){
		start = clock();
	}
	
	
	//FFT(Y)
//	hipfftHandle plan;
//	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		fft[0] = fft[0]+ ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	
	//find Hcon and Hsqrd
	if(timerEn){
		start = clock();
	}
	findHs<< <numOfBlocks,threadsPerBlock-1 >> >(Y, dH, dX, rows, cols);
	hipDeviceSynchronize();
	/*
	if(timerEn){
		finish = clock();
		decode[0] = decode[0]+ ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	*/
	//std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	hipFloatComplex *H = 0;
	H = (hipFloatComplex*)malloc(rows*(cols-1)*sizeof(*H));
	hipMemcpy(H, dH, rows*(cols-1)*sizeof(*dH), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	//std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	
	//H holds Hconj
	//hipMemcpy(H, dH, sizeX*rows, hipMemcpyDeviceToHost);
	/*
	if(timerEn){
		start = clock();
	}
	*/
	//Save |H|^2 into X
	findDistSqrd(H,Hsqrd,rows, cols-1);
	
	
	if(timerEn){
		finish = clock();
		decode[0] = decode[0]+ ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	/*
	std::string file = "Chan_est.dat";
	hipFloatComplex* Yf;
	Yf = (hipFloatComplex*)malloc(rows*(cols-1)*sizeof(*Yf));
	hipMemcpy(Yf, dH, rows*(cols-1)*sizeof(*dH), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	std::cout << "After Chan Est...\n";
	outfile.open(file.c_str(), std::ofstream::binary);
	outfile.write((const char*)Yf, rows*(cols-1)*sizeof(*Yf));
	outfile.close();
	*/
	/*
	memcpy(Yf, Hsqrd, (cols-1)*sizeof(*Hsqrd));
	std::cout << "After Squared...\n";
	file = "Dist_sqrd.dat";
	outfile.open(file.c_str(), std::ofstream::binary);
	outfile.write((const char*)Yf, (cols-1)*sizeof(*Yf));
	outfile.close();
	*/
	
	free(X);
	free(H);
	
	//dH holds H conj
	//dX holds {H^2)	
}


__global__ void doOneSymbol(hipFloatComplex* Y, hipFloatComplex* Hconj, hipFloatComplex* Yf, int rows1, int cols1, int it){
	int rows = rows1;
	int cols= cols1;
    
    //find my work 
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1023
	int row = blockIdx.x;
	/*
	if (row == 0) {
		printf("%d\n",gridDim.x);
	}
	__syncthreads();
	*/
	int j = threadIdx.x;
	//int cp = cols;
	
	
	
//	int c = cols-1;
	
	
	/*
	extern __shared__ hipFloatComplex temp[];
	int tid = threadIdx.x;
	if (j > 0) {
		temp[j-1] = Y[i*c + j];
	}
	__syncthreads();
	
	if (j < cols-1) {
		Yf[i*c + j] = temp[j];
		temp[tid].x = 0;
		temp[tid].y = 0;
	}
	__syncthreads();
	*/
	
//	for (int i = 0; i < rows; i++) {
		
//		for (int j = 0; j < cols-1; j++) {
	__shared__ hipFloatComplex temp[threadsPerBlock-1];
	temp[j] = Y[it*rows*cols + row*(blockDim.x+1) + j + 1];
	__syncthreads();
	Yf[it*rows*(cols-1) + row*blockDim.x + j] = temp[j];
	__syncthreads();
//		}
		
		//memcpy(&Yf[i*(cols-1)], &Y[i*cols+1], (cols-1)* sizeof (*Yf));
//	}
	
	if (j < cols-1) {
	//	Yf[i*c + j] = Y[i*c + j + 1];
		Yf[it*rows*(cols-1) + row*blockDim.x + j] = hipCmulf(Yf[it*rows*(cols-1) + row*blockDim.x + j],Hconj[row*blockDim.x + j]);
	}
	__syncthreads();
	//free(temp);
}

void symbolPreProcess(hipFloatComplex *Y, hipFloatComplex *Hconj, float *Hsqrd,int rows1, int cols1) {
	int rows = rows1;
	int cols= cols1;	
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1023
	
	hipFloatComplex* dY = 0;
	hipMalloc((void**)&dY, rows*cols*(lenOfBuffer-1)*sizeof(*dY));
	hipMemcpy(dY, &Y[rows*cols], rows*cols*(lenOfBuffer-1)*sizeof(*dY), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
//	std::cout << "Symbol " << it << ": " << hipGetErrorString(hipGetLastError()) << std::endl;
	

	{
		std::string file = "Prefix_drop.dat";
		hipFloatComplex *Yf;
		Yf = (hipFloatComplex*)malloc(rows*cols*sizeof(*Yf));
		hipMemcpy(Yf, dY, rows*cols*sizeof(*Yf), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
		std::cout << "\n After Prefix drop:\n";
		for (int j = 0; j < rows*(cols); j = j + cols) {
			cout << "(" << Yf[j].x << ", " << Yf[j].y << ")\n";
		}
		
		outfile.open(file.c_str(), std::ofstream::binary);
		outfile.write((const char*)Yf, rows*(cols)*sizeof(*Yf));
		outfile.close();
		free(Yf);
	}
	
	clock_t start, finish;
	for (int i = 0; i < lenOfBuffer-1; i++) {
		fft[i] = 0;
		if(timerEn){
			start = clock();
		}
		
		//FFT(Y)
		hipfftHandle plan;
		hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
		hipfftExecC2C(plan, (hipfftComplex *)&dY[i*rows*cols], (hipfftComplex *)&dY[i*rows*cols], HIPFFT_FORWARD);
		hipDeviceSynchronize();
		if(timerEn){
			finish = clock();
			fft[i] = fft[i]+ ((float)(finish - start))/(float)CLOCKS_PER_SEC;
		}
	}
	
	{
		std::string file = "FFT_Out.dat";
		hipFloatComplex* Yf;
		Yf = (hipFloatComplex*)malloc(rows*cols*sizeof(*Yf));
		hipMemcpy(Yf, dY, rows*cols*sizeof(*Yf), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
		std::cout << "After FFT...\n";
		outfile.open(file.c_str(), std::ofstream::binary);
		outfile.write((const char*)Yf, rows*(cols)*sizeof(*Yf));
		outfile.close();
		free(Yf);
	}
	
	hipFloatComplex* Yf = 0;
	hipMalloc((void**)&Yf, rows*(cols-1)*(lenOfBuffer-1)* sizeof (*Yf));
	
	if(timerEn){
		start = clock();
	}
	for (int i = 0; i < lenOfBuffer-1; i++) {
		decode[i+1] = 0;
		doOneSymbol<< <numOfBlocks,(threadsPerBlock-1)>> >(dY, Hconj, Yf, rows, cols, i);
	}
	hipDeviceSynchronize();
	/*
	if(timerEn){
		finish = clock();
		decode[it] = ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	*/
	hipMemcpy(Y, Yf, rows*(cols-1)*(lenOfBuffer-1)*sizeof(*Y), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	/*
	if(timerEn){
		start = clock();
	}
	*/
	for (int i = 0; i < lenOfBuffer-1; i++) {
		for(int r=1; r<rows; r++){
			for(int j=0; j<cols-1; j++){
				Y[i*rows*(cols-1) + j]= hipCaddf(Y[i*rows*(cols-1) + j],Y[i*rows*(cols-1) + r*(cols-1)+j]);
			}
		}
		
		//Divide YH* / |H|^2
		for(int j=0; j<cols-1; j++){
			Y[i*rows*(cols-1) + j].x = Y[i*rows*(cols-1) + j].x/Hsqrd[j];
			Y[i*rows*(cols-1) + j].y = Y[i*rows*(cols-1) + j].y/Hsqrd[j];
		}
		shiftOneRow(&Y[i*rows*(cols-1)], cols-1, 0);
	}
	if(timerEn){
		finish = clock();
		decode[1] = ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	
	hipFree(dY);
	hipFree(Yf);
	hipDeviceSynchronize();
}



int main(){
	int rows = numOfRows; // number of vectors
	int cols=dimension;//dimension
	hipSetDevice(0);
	//printf("CUDA LS: \n");
	//printInfo();
	//dY holds symbol with prefix
	hipFloatComplex *dY = 0;
	dY = (hipFloatComplex*)malloc(rows*cols*(lenOfBuffer)* sizeof (*dY));
	
	float *Hsqrd = 0;
	Hsqrd = (float*)malloc((cols-1)*sizeof (*Hsqrd));
	
	//dH (and Hconj) = 16x1023
	hipFloatComplex *dH = 0;
	hipMalloc((void**)&dH, rows*(cols-1)* sizeof (*dH));
	
	//X = 1x1023 -> later can become |H|^2
	hipFloatComplex *dX = 0;
	hipMalloc((void**)&dX, rows*(cols-1)* sizeof (*dX));
	
	/*
	hipFloatComplex *Yf = 0;
	Yf = (hipFloatComplex*)malloc((cols-1)*lenOfBuffer* sizeof (*Yf));
	*/
	//Shared Memory
	string shm_uid = shmemID;
	buffPtr=new ShMemSymBuff(shm_uid, mode);
	
	/*
	hipfftComplex *temp = 0;
	hipMalloc((void**)&temp, rows*cols* sizeof (*temp));
	hipfftHandle plan;
    hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)&temp, (hipfftComplex *)&temp, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	hipFree(temp);
	*/
	
	//Find H* (H conjugate) ->16x1023 and |H|^2 -> 1x1023
	firstVector(dY, dH, dX, Hsqrd, rows, cols);
	//dH holds h conj
	//dX holds |H|^2
	
	for(int i=1; i<numberOfSymbolsToTest; i++){
		if(i==numberOfSymbolsToTest-1){
			//if last one
			buffPtr->readLastSymbolCUDA(dY);
		}
		else{
			buffPtr->readNextSymbolCUDA(dY,i);
			
			if (i == 1) {
				std::string file = "Sym_copy.dat";
//				hipFloatComplex Yf_[rows*(cols+prefix)];
				hipFloatComplex *Yf_;
				Yf_ = (hipFloatComplex*)malloc(rows*(cols)*sizeof(*Yf_));
				memcpy(Yf_, dY, rows*(cols)*sizeof(*Yf_));
				//hipDeviceSynchronize();
				//std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
				std::cout << "Copied back to CPU...\n";
				//printOutArr(Yf_,1,cols+prefix);
					for (int j = 0; j < rows*(cols); j = j + cols) {
						cout << "(" << Yf_[j].x << ", " << Yf_[j].y << ")\n";
					}
				outfile.open(file.c_str(), std::ofstream::binary);
				outfile.write((const char*)Yf_, rows*(cols)*sizeof(*Yf_));
				outfile.close();
			}
			
			
		}
	}
	symbolPreProcess(dY, dH, Hsqrd, rows, cols);
		
	if(testEn){
		//printf("Symbol #%d:\n", i);
		//cuda copy it over
	//	memcpy(Yf, dY, (cols-1)* sizeof (*Yf));
		outfile.open(file.c_str(), std::ofstream::binary | std::ofstream::trunc);
		outfile.write((const char*)dY, (cols-1)*(lenOfBuffer-1)*sizeof(*dY));
		outfile.close();
		//printOutArr(Yf, 1, cols-1);
	}
	
//	free(Yf);
	free(dY);
	hipFree(dH);
	hipFree(dX);
	//delete buffPtr;
	
	if(timerEn) {
	//	printTimes(true);
		storeTimes(false);
	}
	return 0;

}