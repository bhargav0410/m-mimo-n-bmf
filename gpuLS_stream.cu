#include "hip/hip_runtime.h"
#ifndef cudaEn
	#define cudaEn
#endif

//Shared Memory 
#include "ShMemSymBuff_gpu.hpp"
#include "gpuLS.cuh"
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>
#include <csignal>
//#include <boost/thread.hpp>
#define FFT_size dimension
#define cp_size prefix
#define numSymbols lenOfBuffer


/*
	mode:
		= 1 -> master -> creates shared memory 
		= 0 -> slave -> doesn't create the shared memory
*/
 
//!How to Compile:   nvcc ../../examples/gpuLS_full_frame.cu -lcufft -lrt -o gpu -arch=sm_35
// ./gpu

//LS
//Y = 16 x 1024
//X = 1 x 1023
//H = 16 x 1023
using namespace std;

static bool stop_signal_called = false;
void sig_int_handler(int){stop_signal_called = true;}

std::string file = "Output_gpu.dat";
//std::ofstream outfile;

int main(){
	int rows = numOfRows; // number of vectors
	int cols=dimension;//dimension
	device_number = 0;
	hipSetDevice(device_number);
	hipGetDeviceProperties(&gpu->devProp, device_number);
	int maxThreads = gpu->devProp.maxThreadsPerBlock;
	
	//dY holds symbol with prefix
	hipFloatComplex *dY = 0;
	hipMalloc((void**)&dY, rows*(cols-1)*lenOfBuffer* sizeof (*dY));
	
	float *Hsqrd = 0;
	hipMalloc((void**)&Hsqrd, (cols-1)* sizeof (*Hsqrd));
	
	//dH (and Hconj) = 16x1023
	hipFloatComplex *dH = 0;
	hipMalloc((void**)&dH, rows*(cols-1)* sizeof (*dH));
	
	//X = 1x1023 -> later can become |H|^2
	hipFloatComplex *dX = 0;
	hipMalloc((void**)&dX, rows*(cols-1)* sizeof (*dX));
	
	hipFloatComplex *Yf = 0;
	Yf = (hipFloatComplex*)malloc((cols-1)*(lenOfBuffer-1)* sizeof (*Yf));
	
	hipFloatComplex* Y = 0;
	hipMalloc((void**)&Y, rows*cols*lenOfBuffer*sizeof(*Y));
	
	float frameTime;
	
	hipfftHandle plan;
	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	
	gpuLS *gpu = new gpuLS;
	
	//Shared Memory
	string shm_uid = shmemID;
	buffPtr=new ShMemSymBuff(shm_uid, mode);
	std::signal(SIGINT, &sig_int_handler);
	
	gpu->copyPilotToGPU(dX, rows, cols);
	
	
	hipStream_t stream[lenOfBuffer];
	dim3 gridDim(rows,0,0);
	dim3 blockDim(cols,0,0);
	clock_t start, finish;
	clock_t start_total;
	while (not stop_signal_called) {
		start_total = clock();
		&stream[0] = gpu->buffPtr->createStream(0);
		gpu->buffPtr->readNextSymbolCUDA(Y, 0);
		if(timerEn){
			start = clock();
		}
		gpu->batchedFFT(Y, rows, cols, stream[0]);
		if(timerEn){
			finish = clock();
			buffPtr->setFft(((float)(finish - start))/(float)CLOCKS_PER_SEC, 0);
		}
		
		
		if(timerEn){
			start = clock();
		}
		gpu->FindLeastSquaresGPU(, dH, dX, rows, cols, blockDim, gridDim, &stream[0]);
		gpu->FindHsqrdforMRC(dH, Hsqrd, rows, cols, blockDim, gridDim, &stream[0]);
		if(timerEn){
			finish = clock();
			buffPtr->setDecode(((float)(finish - start))/(float)CLOCKS_PER_SEC, 0);
		}
		
		for (int iter = 1; iter < numSymbols; iter++) {
			&stream[iter] = gpu->buffPtr->createStream(iter);
			if(it==numberOfSymbolsToTest-1){
				//if last one
				gpu->buffPtr->readLastSymbolCUDA(&Y[rows*cols*iter]);
			} else {
				gpu->buffPtr->readNextSymbolCUDA(&Y[rows*cols*iter], iter);
			}
			
			if(timerEn){
				start = clock();
			}
			gpu->batchedFFT(&Y[rows*cols*iter], rows, cols, &stream[0]);
			if(timerEn){
				finish = clock();
				buffPtr->setFft(((float)(finish - start))/(float)CLOCKS_PER_SEC, iter);
			}
			
			hipStreamSynchronize(stream[0]);
			
			if(timerEn){
				start = clock();
			}
			gpu->MultiplyWithChannelConj(&Y[rows*cols*iter], dH, &dY[rows*(cols-1)*iter], rows, cols, 1, blockDim, gridDim, &stream[iter]);
			gpu->CombineForMRC(&dY[rows*(cols-1)*iter], Hsqrd, rows, cols, blockDim, gridDim, &stream[iter]);
			gpu->ShiftOneRow(&dY[rows*(cols-1)*iter], cols, 1, blockDim, gridDim, &stream[iter]);
			if(timerEn){
				finish = clock();
				buffPtr->setDecode(((float)(finish - start))/(float)CLOCKS_PER_SEC, iter);
			}
			
			if(testEn){
				//printf("Symbol #%d:\n", i);
				//cuda copy it over
				hipMemcpyAsync(&Yf[(cols-1)*(iter-1)], &dY[rows*(cols-1)*iter], (cols-1)* sizeof (*Yf), cudaMemcpuDeviceToHost, stream[iter]);
				outfile.open(file.c_str(), std::ofstream::binary | std::ofstream::trunc);
				outfile.write((const char*)&Yf[(cols-1)*(iter-1)], (cols-1)*sizeof(*Yf));
				outfile.close();
			}
			gpu->buffPtr->destroyStream(iter);
		}
		gpu->buffPtr->destroyStream(0);
		if(timerEn) {
			gpu->buffPtr->printTimes(true);
			gpu->buffPtr->storeTimes(false);
		}
		
		//Resetting the device and allocating memory for next frame
		hipDeviceReset();
		hipMalloc((void**)&Hsqrd, (cols-1)* sizeof (*Hsqrd));
		hipMalloc((void**)&dH, rows*(cols-1)* sizeof (*dH));
		hipMalloc((void**)&dX, rows*(cols-1)* sizeof (*dX));
		hipMalloc((void**)&Y, rows*cols*lenOfBuffer*sizeof(*Y));
		hipMalloc((void**)&dY, rows*(cols-1)*lenOfBuffer* sizeof (*dY));
		hipfftHandle plan;
		hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
		hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
		hipfftDestroy(plan);
		while ((((float)(clock() - start_total))/(float)CLOCKS_PER_SEC) < 1);
	}
	
	free(Yf);
	hipFree(dY);
	hipFree(Y);
	hipFree(dH);
	hipFree(dX);
	hipFree(Hsqrd);
	delete(gpu);
	hipDeviceReset();
	return 0;

}